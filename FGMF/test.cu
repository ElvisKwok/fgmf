#include "hip/hip_runtime.h"
#include "test.h"

#define lambda 0.05
#define gamma 0.003

typeRate computeRMSE_new(sRateNode *rateNodeArray, typeRate *matrixP, typeRate *matrixQ, int K, int NNZ)
{
    typeRate rmse = 0;

    for(int i = 0; i < NNZ; ++i)
    {
        typeRate rate = rateNodeArray[i].rate;
        int userIdx = rateNodeArray[i].u - 1;
        int itemIdx = rateNodeArray[i].i - 1;
        typeRate predictRate = 0;

        for(int k = 0; k < K; ++k)
        {
            predictRate += (*(matrixP + userIdx * K + k)) * (*(matrixQ + itemIdx * K + k));
        }

        rmse += pow((rate - predictRate), 2);
    }

    return sqrt(rmse / NNZ);
}

__device__ void sgdUpdateNew(typeRate rate, typeRate *matrixP, typeRate *matrixQ, int userIdx, int itemIdx, int K)
{
    typeRate predictRate = 0;

    for(int k = 0; k < K; ++k)
    {
        predictRate += (*(matrixP + userIdx * K + k)) * (*(matrixQ + itemIdx * K + k));
    }

    typeRate err = rate - predictRate;

    for(int k = 0; k < K; ++k)
    {
        (*(matrixP + userIdx * K + k)) += gamma * (2 * err * (*(matrixQ + itemIdx * K + k)) - lambda * (*(matrixP + userIdx * K + k)));
        (*(matrixQ + itemIdx * K + k)) += gamma * (2 * err * (*(matrixP + userIdx * K + k)) - lambda * (*(matrixQ + itemIdx * K + k)));
    }
}

__global__ void sgd_kernelNew(sRateNode *d_rateNodeArray, typeRate *d_matrixP, typeRate *d_matrixQ, int K,
                              sWorkset *d_worksetArray, sWorkseg *d_mWorkseg, int *d_mPattern, int s,
                              int subBlockNumL, int subBlockLen)
{
    int tid = threadIdx.x;
    int tbid = blockIdx.x;

    if(tbid > subBlockNumL)
        return;

    int bid = *(d_mPattern + s * subBlockNumL + tbid);

    if(bid == -1 || d_worksetArray[bid].beg == d_worksetArray[bid].end)
        return;

    for(int tag = 0; tag < subBlockLen; ++tag)
    {
        int from = (*(d_mWorkseg + tbid * subBlockLen + tag)).from;
        int to = (*(d_mWorkseg + tbid * subBlockLen + tag)).to;

        for(int iRate = from + tid; iRate < to; iRate += blockDim.x)
        {
            typeRate rate = d_rateNodeArray[iRate].rate;
            int userIdx = d_rateNodeArray[iRate].u - 1;
            int itemIdx = d_rateNodeArray[iRate].i - 1;
            sgdUpdateNew(rate, d_matrixP, d_matrixQ, userIdx, itemIdx, K);
        }

        __syncthreads();
    }
}


void callKernel(sRateNode *rateNodeArray, typeRate *matrixP, typeRate *matrixQ, int M, int N, int K,
                sWorkset *worksetArray, sWorkseg *mWorkseg, int *mPattern,
                int subBlockNumL, int subBlockLen, int NNZ)
{
    sRateNode *d_rateNodeArray;
    typeRate *d_matrixP;
    typeRate *d_matrixQ;
    sWorkset *d_worksetArray;
    sWorkseg *d_mWorkseg;
    int *d_mPattern;
    int subBlockNum = subBlockNumL * subBlockNumL;
    hipMalloc((void**)&d_rateNodeArray, NNZ * sizeof(sRateNode));
    hipMalloc((void**)&d_matrixP, M * K * sizeof(typeRate));
    hipMalloc((void**)&d_matrixQ, N * K * sizeof(typeRate));
    hipMalloc((void**)&d_worksetArray, subBlockNum * sizeof(sWorkset));
    hipMalloc((void**)&d_mWorkseg, subBlockNum * subBlockLen * sizeof(sWorkseg));
    hipMalloc((void**)&d_mPattern, subBlockNum * sizeof(int));
    hipMemcpy(d_rateNodeArray, rateNodeArray, NNZ * sizeof(sRateNode), hipMemcpyHostToDevice);
    hipMemcpy(d_matrixP, matrixP, M * K * sizeof(typeRate), hipMemcpyHostToDevice);
    hipMemcpy(d_matrixQ, matrixQ, N * K * sizeof(typeRate), hipMemcpyHostToDevice);
    hipMemcpy(d_worksetArray, worksetArray, subBlockNum * sizeof(sWorkset), hipMemcpyHostToDevice);
    hipMemcpy(d_mWorkseg, mWorkseg, subBlockNum * subBlockLen * sizeof(sWorkseg), hipMemcpyHostToDevice);
    hipMemcpy(d_mPattern, mPattern, subBlockNum * sizeof(int), hipMemcpyHostToDevice);

    for(int iter = 0; iter < MAX_ITER; ++iter)
    {
        auto start = system_clock::now();

        for(int s = 0; s < subBlockNumL; ++s)
        {
            sgd_kernelNew << < subBlockNumL, 1024 >> >(
                              d_rateNodeArray,
                              d_matrixP,
                              d_matrixQ,
                              K,
                              d_worksetArray,
                              d_mWorkseg,
                              d_mPattern,
                              s,
                              subBlockNumL,
                              subBlockLen
                          );
            hipDeviceSynchronize();
        }

        auto end = system_clock::now();
        auto duration = duration_cast<microseconds>(end - start);
        cout << "it takes iter " << iter << "\t\t" << double(duration.count()) * microseconds::period::num / microseconds::period::den << " seconds" << endl;
        hipMemcpy(matrixP, d_matrixP, M * K * sizeof(typeRate), hipMemcpyDeviceToHost);
        hipMemcpy(matrixQ, d_matrixQ, N * K * sizeof(typeRate), hipMemcpyDeviceToHost);
        cout << "RMSE: " << computeRMSE_new(rateNodeArray, matrixP, matrixQ, K, NNZ) << endl;
    }

    hipFree(d_rateNodeArray);
    hipFree(d_matrixP);
    hipFree(d_matrixQ);
    hipFree(d_worksetArray);
    hipFree(d_mWorkseg);
    hipFree(d_mPattern);
}
